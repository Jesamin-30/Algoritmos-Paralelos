#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include "opencv2/highgui/highgui.hpp"
#include <iostream>

#include "pixel.h"

#define blur_size 3

__global__
void blur_kernel(pixel *in, pixel *out, unsigned w, unsigned h){
    unsigned col = threadIdx.x+(blockIdx.x*blockDim.x);
	unsigned row = threadIdx.y+(blockIdx.y*blockDim.y);
    
    if(col<w and row<h){
    	float valr = 0.0;
        float valg = 0.0;
        float valb = 0.0;
        float size = 0;

        int crow, ccol;
        for(int brow=-blur_size; brow<blur_size+1; brow++){
            for(int bcol=blur_size; bcol<blur_size+1; bcol++){
                crow = row+brow;
                ccol = col+bcol;

                if(crow>-1 and crow<h && ccol>-1 and ccol<w){
                    valr += in[(crow*w)+ccol].get_r();
                    valg += in[(crow*w)+ccol].get_g();
                    valb += in[(crow*w)+ccol].get_b();
                    size+=1;
                }
            }
        }

        out[(row*w)+col].set_r(valr/size);
        out[(row*w)+col].set_g(valg/size);
        out[(row*w)+col].set_b(valb/size);
    }
}

void kblur(pixel *h_in, pixel *h_gray, unsigned col, unsigned row){
    unsigned size = col*row*sizeof(pixel);

    pixel *d_in;
    pixel *d_out;

    hipMalloc((void **)&d_in, size);
	hipMalloc((void **)&d_out, size);

	hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

	unsigned block = 30;
	dim3 dimGrid(ceil(col/block), ceil(row/block), 1);
	dim3 dimBlock(block, block, 1);
	blur_kernel<<<dimGrid, dimBlock>>>(d_in, d_out, col, row);

	hipMemcpy(h_gray, d_out, size, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
}



int main(int argc, char const *argv[]){

	cv::Mat src_in = cv::imread("lena.png", cv::IMREAD_COLOR);
	if(src_in.empty()){
		std::cout << "Error : Image cannot be loaded..!!n";
		return 0;
	}

	cv::namedWindow("window_in", CV_WINDOW_NORMAL);
	cv::imshow("window_in", src_in);

	unsigned cols = src_in.cols;
	unsigned rows = src_in.rows;

	std::cout << "cols: " << cols << "\n";
	std::cout << "rows: " << rows << "\n";

	pixel *in = new pixel[cols*rows];
	pixel *blur = new pixel[cols*rows];

	unsigned i, j; //tmp;
	for(i=0; i<rows; i++){
		for(j=0; j<cols; j++){
		    in[(i*cols)+j].set_r((float)src_in.at<cv::Vec3b>(i, j)[0]);
		    in[(i*cols)+j].set_g((float)src_in.at<cv::Vec3b>(i, j)[1]);
		    in[(i*cols)+j].set_b((float)src_in.at<cv::Vec3b>(i, j)[2]);
		}
	}	

	kblur(in, blur, cols, rows);

	cv::Mat src_out(rows, cols, CV_8UC3, cv::Scalar(0,0,0));
	for(i=0; i<rows; i++){
		for(j=0; j<cols; j++){
		    src_out.at<cv::Vec3b>(i, j)[0] = blur[(i*cols)+j].get_r();
		    src_out.at<cv::Vec3b>(i, j)[1] = blur[(i*cols)+j].get_g();
		    src_out.at<cv::Vec3b>(i, j)[2] = blur[(i*cols)+j].get_b();
		}
	}

	cv::namedWindow("window_blur", CV_WINDOW_NORMAL);
	cv::imshow("window_blur", src_out);

	cv::waitKey(0);
	cv::destroyWindow("window_in");
	cv::destroyWindow("window_blur");

	delete in;
	delete blur;
	return 0;
}