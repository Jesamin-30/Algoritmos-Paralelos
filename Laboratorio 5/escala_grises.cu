#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include "opencv2/highgui/highgui.hpp"
#include <iostream>
#include "pixel.h"

__global__
void gray_scale_kernel(pixel *in, float *gray, unsigned col, unsigned row){
    unsigned c = threadIdx.x+(blockIdx.x*blockDim.x);
	unsigned r = threadIdx.y+(blockIdx.y*blockDim.y);
    
    if(c<col and r<row){    	
    	unsigned id = (r*col)+c;
    	gray[id] = (0.299*in[id].get_r())+
                    (0.587*in[id].get_g())+
                    (0.114*in[id].get_b());
    }
}

void gray_scale(pixel *in, float *h_out, unsigned col, unsigned row){
    unsigned msize = col*row*sizeof(pixel);     // pixel vector
    unsigned rsize = col*row*sizeof(float);     // float vector

    pixel *d_in;
    float *d_out;

    hipMalloc((void **)&d_in, msize);
	hipMalloc((void **)&d_out, rsize);

	hipMemcpy(d_in, in, msize, hipMemcpyHostToDevice);

	unsigned block = 16;
	dim3 dimGrid(ceil(col/block), ceil(row/block), 1);
	dim3 dimBlock(block, block, 1);
	gray_scale_kernel<<<dimGrid, dimBlock>>>(d_in, d_out, col, row);

	hipMemcpy(h_out, d_out, rsize, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
}


int main(int argc, char const *argv[]){

	cv::Mat src_in = cv::imread("lena.png", cv::IMREAD_COLOR);
	if(src_in.empty()){
		std::cout << "Error : Image cannot be loaded..!!n";
		return 0;
	}

	cv::namedWindow("window_in", CV_WINDOW_NORMAL);
	cv::imshow("window_in", src_in);

	unsigned cols = src_in.cols;
	unsigned rows = src_in.rows;

	std::cout << "cols: " << cols << "\n";
	std::cout << "rows: " << rows << "\n";

	pixel *in = new pixel[cols*rows];
	float *out = new float[cols*rows];


	unsigned tmp, i, j;
	for(i=0; i<rows; i++){
		for(j=0; j<cols; j++){
		    in[(i*cols)+j].set_r((float)src_in.at<cv::Vec3b>(i, j)[0]);
		    in[(i*cols)+j].set_g((float)src_in.at<cv::Vec3b>(i, j)[1]);
		    in[(i*cols)+j].set_b((float)src_in.at<cv::Vec3b>(i, j)[2]);
		}
	}

	gray_scale(in, out, cols, rows);

	cv::Mat src_out(rows,cols, CV_8UC3, cv::Scalar(0,0,0));	
	for(i=0; i<rows; i++){
		for(j=0; j<cols; j++){
			tmp = (i*cols)+j;
			src_out.at<cv::Vec3b>(i, j)[0] = out[tmp];
			src_out.at<cv::Vec3b>(i, j)[1] = out[tmp];
			src_out.at<cv::Vec3b>(i, j)[2] = out[tmp];
		}
	}

	cv::namedWindow("window_out", CV_WINDOW_NORMAL);
	cv::imshow("window_out", src_out);

	cv::waitKey(0);
	cv::destroyWindow("window_in");
	cv::destroyWindow("window_out");

	delete in;
	delete out;
	return 0;
}