
#include <hip/hip_runtime.h>
#include <iostream>
#include <unistd.h>
#include <sys/time.h>

#define tile_width 32

__global__
void normal_square_matrix_mult_kernel(int *m, int *n, int *p, unsigned width){
	unsigned col = threadIdx.x+(blockIdx.x*blockDim.x);
	unsigned row = threadIdx.y+(blockIdx.y*blockDim.y);

	if(col<width and row<width){
		int pvalue = 0;
		for(unsigned k=0; k<width; k++){
			pvalue += m[(row*width)+k]*n[(k*width)+col];
		}
		p[(row*width)+col] = pvalue;
	}
}

__global__
void tile_square_matrix_mult_kernel(int *m, int *n, int *p, unsigned width){
	__shared__ int mds[tile_width][tile_width];
	__shared__ int nds[tile_width][tile_width];

	unsigned bx = blockIdx.x;
	unsigned by = blockIdx.y;

	unsigned tx = threadIdx.x;
	unsigned ty = threadIdx.y;

	unsigned row = (by*tile_width)+ty;	
	unsigned col = (bx*tile_width)+tx;

	int pvalue = 0;
	unsigned ph, k;
	for(ph=0; ph<width/tile_width; ph++){
		mds[ty][tx] = m[(row*width)+(ph*tile_width) + tx];
		nds[ty][tx] = n[(((ph*tile_width)+ty)*width) + col];
		__syncthreads();

		for(k=0; k<tile_width; k++){
			pvalue += mds[ty][k]*nds[k][tx];
		}
		__syncthreads();
	}

	p[(row*width)+col] = pvalue;
}

void square_matrix_mult(int *m, int *n, int *p, unsigned width, unsigned block, char type){
	unsigned size = width*width*sizeof(int);
	int *d_m, *d_n, *d_p;

	hipMalloc((void **)&d_m, size);
	hipMalloc((void **)&d_n, size);
	hipMalloc((void **)&d_p, size);

	hipMemcpy(d_m, m, size, hipMemcpyHostToDevice);	
	hipMemcpy(d_n, n, size, hipMemcpyHostToDevice);	

	dim3 dimGrid(ceil(width/block), ceil(width/block), 1);
	dim3 dimBlock(block, block, 1);

	switch(type){
		case 'n':
			normal_square_matrix_mult_kernel<<<dimGrid, dimBlock>>>(d_m, d_n, d_p, width);
			break;
		case 't':
			tile_square_matrix_mult_kernel<<<dimGrid, dimBlock>>>(d_m, d_n, d_p, width);
			break;
		default:
			std::cout << "  type [0]-[1]!!!";
			break;
	}	

	hipMemcpy(p, d_p, size, hipMemcpyDeviceToHost);

	hipFree(d_m);
	hipFree(d_n);
	hipFree(d_p);
}


int main(int argc, char const *argv[]){	
	unsigned width = 1024;
	unsigned block = 32;

	unsigned ntotal = width*width;	

	int *h_m = new int[ntotal];
	int *h_n = new int[ntotal];
	int *h_p = new int[ntotal];
	int *h_r = new int[ntotal];

	unsigned i, j;
	for(i=0; i<width; i++){
		for(j=0; j<width; j++){
		    h_m[(i*width)+j] = j;
		    h_n[(i*width)+j] = j;
		    h_p[(i*width)+j] = 0;
		    h_r[(i*width)+j] = 0;
		}
	}
		
	struct timeval ti, tf;
	double time;

	gettimeofday(&ti, NULL);

	square_matrix_mult(h_m, h_n, h_p, width, block, 'n');

	gettimeofday(&tf, NULL);
	
	time = (tf.tv_sec - ti.tv_sec)*1000 + (tf.tv_usec - ti.tv_usec)/1000;
	printf("[%ix%i] memoria global: %.8lf s\n", width, width, time/1000);
	// print_matrix(h_p, width, width);

	gettimeofday(&ti, NULL);
		square_matrix_mult(h_m, h_n, h_r, width, block, 't');
	gettimeofday(&tf, NULL);
	time = (tf.tv_sec - ti.tv_sec)*1000 + (tf.tv_usec - ti.tv_usec)/1000;
	printf("[%ix%i] memoria compartida: %.8lf s\n", width, width, time/1000);
	// print_matrix(h_r, width, width);

	delete h_m;
	delete h_n;
	delete h_p;
	delete h_r;

	return 0;
}